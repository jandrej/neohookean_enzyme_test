#include "hip/hip_runtime.h"
#include "cuda_kernels.hpp"

template<typename return_type, typename... Args>
__device__ return_type __enzyme_fwddiff(Args...);

__device__ int enzyme_dup;
__device__ int enzyme_dupnoneed;
__device__ int enzyme_out;
__device__ int enzyme_const;

namespace cuda_kernels {

template <int dim, int num_el, int num_qp> 
__global__ void mock_fem_loop_kernel(
  tensor<double, dim, dim> &du_dx,
  tensor<double, dim, dim> &perturbation,
  double &C1,
  double &D1,
  tensor<double, dim, dim> &sigma,
  tensor<double, dim, dim> &dsigma) {
  
  for (int e = 0; e < num_el; e++) {
    for (int q = 0; q < num_qp; q++) {
      __enzyme_fwddiff<void>(stress_calculation,
                        enzyme_dup, &du_dx, &perturbation,
                        enzyme_const, C1,
                        enzyme_const, D1,
                        enzyme_dupnoneed, &sigma, &dsigma);
    }
  }
}

void mock_fem_loop() {
  double epsilon = 1.0e-8;
  tensor<double, 3, 3> du_dx = {{{0.2, 0.4, -0.1}, {0.2, 0.1, 0.3}, {0.01, -0.2, 0.3}}};
  tensor<double, 3, 3> perturbation = {{{1.0, 0.2, 0.8}, {2.0, 0.1, 0.3}, {0.4, 0.2, 0.7}}};

  double C1 = 100.0;
  double D1 = 50.0;


  tensor<double, 3, 3> sigma{};
  tensor<double, 3, 3> dsigma{};

  mock_fem_loop_kernel<3, 1024, 9><<<1,1>>>(du_dx, perturbation, C1, D1, sigma, dsigma);
}

};